#include "hip/hip_runtime.h"
#include <linearRegression.cuh>
#include <random>
#include <lr_kernels.cuh>
#include <hipcub/hipcub.hpp>

#define CUDA_CHECK(call) do {                                  \
    hipError_t err = call;                                    \
    if (err != hipSuccess) {                                  \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n",    \
                __FILE__, __LINE__, hipGetErrorString(err));  \
        exit(err);                                             \
    }                                                          \
} while(0)

void linearRegression::fit(float *X_train, float *y_train, int N, int d, float p, int epochs) {
    this->X_train = X_train, this->y_train = y_train, this->p = p;
    this->N = N, this->d = d;
    float *g = (float *)calloc(sizeof(float), d); // gradients

    this->weights = (float *)malloc(sizeof(float) * d);
    std::mt19937 rng(std::random_device{}());
    std::normal_distribution<float> dist(0.0f, 1.0f/std::sqrt(d));
    bias = dist(rng);
    for(int i = 0; i < d; ++i)
        weights[i] = dist(rng);

    hipMalloc(&d_X, sizeof(float) * N * d);
    hipMalloc(&d_y, sizeof(float) * N);
    hipMalloc(&d_w, sizeof(float) * d);
    hipMemcpy(d_X, X_train, sizeof(float) * N * d, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y_train, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_w, weights, sizeof(float) * d, hipMemcpyHostToDevice);

    int e, i;

    for(e = 1; e <= epochs; e++){
        float g0 = compute_g(g);
        this->bias -= this->p * g0;
        
        #pragma unroll 5
        for(i = 0; i < d; i++) weights[i] -= p * g[i];
        hipMemcpy(d_w, weights, sizeof(float) * d, hipMemcpyHostToDevice);

        // if (e % 100 == 0) {
        //     printf("Epoch - %d\n", e);

        //     for(i = 0; i < d; i++) printf("weight[%d]=%f\n", i, weights[i]);
        //     printf("bias=%f\n\n", bias);
        // }
    }

    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_w);

    printf("\nFinal Weights and Bias:\n");
    printf("Bias - %f\n", bias);
    for(i = 0; i < d; i++) printf("Weight[%d]=%f\n", i + 1, weights[i]);
    printf("\n");
}

float *linearRegression::predict(float *X_test, float *y_test, int N, int d, float *MSE){
    float *preds = (float *)malloc(sizeof(float) * N);

    float *d_mse_r, *d_preds;
    hipMalloc(&d_mse_r, sizeof(float) * N);
    hipMalloc(&d_preds, sizeof(float) * N);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    hipMalloc(&d_X, sizeof(float) * N * d);
    hipMalloc(&d_y, sizeof(float) * N);
    hipMalloc(&d_w, sizeof(float) * d);
    hipMemcpy(d_X, X_test, sizeof(float) * N * d, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y_test, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_w, weights, sizeof(float) * d, hipMemcpyHostToDevice);

    predict_kernel<<<blocks, threads>>>(d_X, d_y, d_w, d_preds, d_mse_r, N, d, bias);

    hipDeviceSynchronize();

    // compute mse
    float *d_mse = nullptr;
    hipMalloc(&d_mse, sizeof(float));

    void * tmp = nullptr;
    size_t tmp_bytes = 0;

    hipcub::DeviceReduce::Sum(tmp,tmp_bytes, d_mse_r, d_mse, N);

    hipMalloc(&tmp, tmp_bytes);
    hipcub::DeviceReduce::Sum(tmp, tmp_bytes, d_mse_r, d_mse, N);

    hipFree(tmp);
    hipMemcpy(MSE, d_mse, sizeof(float), hipMemcpyDeviceToHost);

    hipMemcpy(preds, d_preds, sizeof(float) * N, hipMemcpyDeviceToHost);

    hipFree(d_X);
    hipFree(d_y);
    hipFree(d_w);
    hipFree(d_mse_r);
    hipFree(d_preds);
    hipFree(d_mse);

    *MSE /= N;

    return preds;
}

float linearRegression::compute_g(float *g_array){
    float *d_residuals = nullptr;
    hipMalloc(&d_residuals, N * sizeof(float));

    // compute g0
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    compute_residuals<<<blocks, threads>>>(d_X, d_y, d_w, d_residuals, bias, N, d);

    hipDeviceSynchronize();

    // sum reduction
    float *d_g0 = nullptr;
    hipMalloc(&d_g0, sizeof(float));

    void * tmp = nullptr;
    size_t tmp_bytes = 0;

    hipcub::DeviceReduce::Sum(tmp,tmp_bytes, d_residuals, d_g0, N);

    hipMalloc(&tmp, tmp_bytes);
    hipcub::DeviceReduce::Sum(tmp, tmp_bytes, d_residuals, d_g0, N);

    hipFree(tmp);
    
    float g0 = 0.0f;
    hipMemcpy(&g0, d_g0, sizeof(float), hipMemcpyDeviceToHost);
    // compute rest of gradients
    blocks = d;
    size_t shmem = sizeof(float) * threads;
    float *d_g;
    hipMalloc(&d_g, sizeof(float) * N);

    compute_g_kernel<<<blocks, threads, shmem>>>(d_X, d_residuals, d_g, N, d);
    hipDeviceSynchronize();
    hipMemcpy(g_array, d_g, sizeof(float) * d, hipMemcpyDeviceToHost);

    hipFree(d_residuals);
    hipFree(d_g0);

    return g0;
}