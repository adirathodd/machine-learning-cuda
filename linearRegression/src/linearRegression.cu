#include <linearRegression.cuh>
#include <random>
#include <hip/hip_runtime.h>
#include <lr_kernels.cuh>
#include <cstdio>
#include <cstdlib>
#define CUDA_CHECK(call) do {                                  \
    hipError_t err = call;                                    \
    if (err != hipSuccess) {                                  \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n",    \
                __FILE__, __LINE__, hipGetErrorString(err));  \
        exit(err);                                             \
    }                                                          \
} while(0)

void linearRegression::fit(vector<float> X_train, vector<float> y_train, int numRows, int numCols, float p, int epochs) {
    this->X_train = X_train, this->y_train = y_train, this->p = p;
    float *d_X_train, *d_y_train, *d_weights;

    // initialize weights and bias
    std::mt19937 rng(42);
    this->bias = std::uniform_real_distribution<float>(-1.0f, 1.0f)(rng);
    this->weights = new float[numCols];
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    
    int k = 100;
    int numBatches = numRows / k;
    if (numRows % k != 0) numBatches++;

    CUDA_CHECK(hipMalloc((void**)&d_X_train, numRows * numCols * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_y_train, numRows * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_weights, numCols * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_X_train, X_train.data(), numRows * numCols * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y_train, y_train.data(), numRows * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, weights, numCols * sizeof(float), hipMemcpyHostToDevice));

    float g[numCols];

    for(int e = 0; e < epochs; e++) {

        for (int i = 0; i < numCols; i++) {
            printf("Weight %d: %f\n", i+1, weights[i]);
        }
        printf("-------------------------------------\n\n");
        
        for(int t = 0; t < numBatches; t++) {
            // compute g_0
            float g_0 = compute_g0(numCols, t, k, d_X_train, d_y_train, d_weights);

            //compute loss for each feature
            for(int i = 0; i < numCols; i++){
                g[i] = compute_gi(numCols, i, t, k, d_X_train, d_y_train, d_weights);
            }

            //update bias
            this->bias = this->bias - (this->p * g_0);

            //update weights
            for(int i = 0; i < numCols; i++) this->weights[i] = this->weights[i] - (this->p * g[i]);
            CUDA_CHECK(hipMemcpy(d_weights, weights, numCols * sizeof(float), hipMemcpyHostToDevice));

        }
        
    }

    printf("Final Bias: %f\n", bias);
    for (int i = 0; i < numCols; i++) {
        printf("Final Weight %d: %f\n", i+1, weights[i]);
    }

    return;
}

float linearRegression::predict(vector<float> row) {
    float prediction = 0.0f;
    for (int i = 0; i < row.size(); i++) {
        prediction += row[i] * weights[i];
    }
    return prediction;
}

float linearRegression::compute_g0(int numCols, int t, int k, float *d_X_train, float *d_y_train, float *d_weights) {
    float *d_g0;
    CUDA_CHECK(hipMalloc((void**)&d_g0, k * sizeof(float)));

    int threadsPerBlock = 32;
    int blocksPerGrid = (k + threadsPerBlock - 1) / threadsPerBlock;
    computeG0<<<blocksPerGrid, threadsPerBlock>>>(numCols, t, k, bias,
                                                d_X_train, d_y_train, d_weights, d_g0);

    CUDA_CHECK(hipDeviceSynchronize());

    int reduceThreads = 32;
    int reduceBlocks = (k + reduceThreads * 2 - 1) / (reduceThreads * 2);
    size_t sharedMemSize = reduceThreads * sizeof(float);
    float *d_partialSums;
    CUDA_CHECK(hipMalloc((void**)&d_partialSums, reduceBlocks * sizeof(float)));

    reduceSum<<<reduceBlocks, reduceThreads, sharedMemSize>>>(d_g0, d_partialSums, k);
    CUDA_CHECK(hipDeviceSynchronize());

    int s = reduceBlocks;
    while (s > 1) {
        int threads = (s < reduceThreads * 2 ? s / 2 : reduceThreads);
        int blocks = (s + threads * 2 - 1) / (threads * 2);
        reduceSum<<<blocks, threads, threads * sizeof(float)>>>(d_partialSums, d_partialSums, s);
        CUDA_CHECK(hipDeviceSynchronize());
        s = blocks;
    }

    float h_g0;
    CUDA_CHECK(hipMemcpy(&h_g0, d_partialSums, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_g0));
    CUDA_CHECK(hipFree(d_partialSums));

    return h_g0;
}

float linearRegression::compute_gi(int numCols, int i, int t, int k, float *d_X_train, float *d_y_train, float *d_weights) {
    float *d_g_i;
    CUDA_CHECK(hipMalloc((void**)&d_g_i, k * sizeof(float)));

    int threadsPerBlock = 256;
    int blocksPerGrid = (k + threadsPerBlock - 1) / threadsPerBlock;

    // d_g_i[j] = (bias + dot(X[j], w) - y[j]) * X[j * numCols + i]
    computeGi<<<blocksPerGrid, threadsPerBlock>>>(numCols, i, t, k, bias,
                    d_X_train, d_y_train, d_weights, d_g_i);
    CUDA_CHECK(hipDeviceSynchronize());

    int reduceThreads = 256;
    int reduceBlocks = (k + reduceThreads * 2 - 1) / (reduceThreads * 2);
    size_t sharedMemSize = reduceThreads * sizeof(float);
    float *d_partialSums;
    CUDA_CHECK(hipMalloc((void**)&d_partialSums, reduceBlocks * sizeof(float)));

    reduceSum<<<reduceBlocks, reduceThreads, sharedMemSize>>>(d_g_i, d_partialSums, k);
    CUDA_CHECK(hipDeviceSynchronize());

    int s = reduceBlocks;
    while (s > 1) {
        int threads = (s < reduceThreads * 2 ? s / 2 : reduceThreads);
        int blocks = (s + threads * 2 - 1) / (threads * 2);
        reduceSum<<<blocks, threads, threads * sizeof(float)>>>(d_partialSums, d_partialSums, s);
        CUDA_CHECK(hipDeviceSynchronize());
        s = blocks;
    }

    float h_g_i;
    CUDA_CHECK(hipMemcpy(&h_g_i, d_partialSums, sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipFree(d_g_i));
    CUDA_CHECK(hipFree(d_partialSums));

    return h_g_i;
}
